#include <stdio.h>
#include <hip/hip_runtime.h>

// Kernel de CUDA (se ejecuta en la GPU)
__global__ void sumaVectores(float* A, float* B, float* C, int n) {
    int i = threadIdx.x;  // Obtiene el índice del hilo actual
    if (i < n) {
        C[i] = A[i] + B[i];  // Realiza la suma elemento por elemento
    }
}

int main() {
    const int N = 5;  // Tamaño de los vectores
    float h_A[N], h_B[N], h_C[N];  // Vectores en el host (CPU)
    float *d_A, *d_B, *d_C;       // Punteros para device (GPU)

    // Inicializar vectores de entrada
    for (int i = 0; i < N; i++) {
        h_A[i] = i;
        h_B[i] = i * 2;
    }

    // 1. Reservar memoria en la GPU
    hipMalloc(&d_A, N * sizeof(float));
    hipMalloc(&d_B, N * sizeof(float));
    hipMalloc(&d_C, N * sizeof(float));

    // 2. Copiar datos del host al device
    hipMemcpy(d_A, h_A, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * sizeof(float), hipMemcpyHostToDevice);

    // 3. Lanzar el kernel con N hilos (1 bloque de N hilos)
    sumaVectores<<<1, N>>>(d_A, d_B, d_C, N);

    // 4. Copiar resultado de vuelta al host
    hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    // 5. Liberar memoria de la GPU
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Mostrar resultados
    printf("Vector A: ");
    for (int i = 0; i < N; i++) printf("%.2f  ", h_A[i]);
    
    printf("\nVector B: ");
    for (int i = 0; i < N; i++) printf("%.2f  ", h_B[i]);
    
    printf("\nResultado: ");
    for (int i = 0; i < N; i++) printf("%.2f  ", h_C[i]);

    return 0;
}
